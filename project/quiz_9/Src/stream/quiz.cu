#include "hip/hip_runtime.h"
#include <stdio.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


// DO NOT change the kernel function
__global__ void vector_add(int *a, int *b, int *c)
{
// DO NOT change the kernel function
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}


#define N (2048*2048)
#define THREADS_PER_BLOCK 128

int main()
{
    int *a, *b, *c, *golden;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof( int );

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );
	golden = (int *)malloc(size);

	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		golden[i] = a[i] + b[i];
		c[i] = 0;
	}


	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

	vector_add <<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c );

	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );

	bool pass = true;
	for (int i = 0; i < N; i++) {
		if (golden[i] != c[i])
			pass = false;
	}
	
	if (pass)
		printf("PASS\n");
	else
		printf("FAIL\n");

	printf("print your name and id\n");

	free(a);
	free(b);
	free(c);
	free(golden);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} 
